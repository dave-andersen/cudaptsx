#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2014 David G. Andersen
 * This code is licensed under the Apache 2.0 license and may be used or re-used
 * in accordance with its terms.
 */

#include <inttypes.h>
#include <stdio.h>
#include "gpuhash.h"
#include "hip/hip_runtime.h"
#include "defs.h"
#include "shabits.h"
//#include <thrust/sort.h>

__device__ void sha512_block(uint64_t H[8], const uint64_t data[5]);
__global__ void search_sha512_kernel(const __restrict__ uint64_t *dev_data, __restrict__ uint64_t *dev_hashes, __restrict__ uint32_t *dev_countbits);
__global__ void filter_sha512_kernel(__restrict__ uint64_t *dev_hashes, const __restrict__ uint32_t *dev_countbits, hipTextureObject_t dc_as_tex);
__global__ void filter_and_rewrite_sha512_kernel(__restrict__ uint64_t *dev_hashes, const __restrict__ uint32_t *dev_countbits, hipTextureObject_t dc_as_tex, __restrict__ uint64_t *dev_results);
__global__ void populate_filter_kernel(__restrict__ uint64_t *dev_hashes, __restrict__ uint32_t *dev_countbits);


/* Empty constructor, please call Initialize */
GPUHasher::GPUHasher(int gpu_device_id) {
  device_id = gpu_device_id;
}

/* UGGGGGGGGGH temporary hack fix me  - put in an opaque in the .h */
hipTextureObject_t dc_as_tex = 0;

int GPUHasher::Initialize() {
  hipError_t error;
  
  error = hipSetDevice(device_id);
  if (error != hipSuccess) {
    fprintf(stderr, "Could not attach to CUDA device %d: %d\n", device_id, error);
    exit(-1);
  }

  hipStream_t *streamptr = (hipStream_t *)opaqueStream_t;
  error = hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

  size_t free, total;
  hipMemGetInfo(&free, &total);
  printf("Initializing.  Device has %ld free of %ld total bytes of memory\n", free, total);

  error = hipMalloc((void **)&dev_data, sizeof(uint64_t)*16);
  if (error != hipSuccess) {
    fprintf(stderr, "Could not malloc dev_data (%d)\n", error);
    exit(-1);
    return -1;
  }

  hipStreamCreate(streamptr);

#define MOMENTUM_N_HASHES (1<<26)
  /* Note:  This is the allocation size.  We can only use
   * one less than this because each countbit entry uses two bits. */
#define NUM_COUNTBITS_POWER 31
#define GPU_COUNTBITS_SLOTS_POWER (NUM_COUNTBITS_POWER-1)
#define NUM_COUNTBITS_WORDS (1<<(NUM_COUNTBITS_POWER-5))

  error = hipMalloc((void **)&dev_hashes, sizeof(uint64_t)*MOMENTUM_N_HASHES);
  if (error != hipSuccess) {
    fprintf(stderr, "Could not malloc dev_data (%d)\n", error);
    return -1;
  }

  error = hipMalloc((void **)&dev_countbits, sizeof(uint32_t)*NUM_COUNTBITS_WORDS);
  if (error != hipSuccess) {
    fprintf(stderr, "Could not malloc dev_data (%d)\n", error);
    exit(-1);
    return -1;
  }

  /* Results holds any maybe-colliding keys */
  error = hipMalloc((void **)&dev_results, sizeof(uint64_t)*GPUHasher::N_RESULTS);
  if (error != hipSuccess) {
    fprintf(stderr, "Could not malloc dev_data (%d)\n", error);
    exit(-1);
    return -1;
  }

  hipFuncSetCacheConfig(reinterpret_cast<const void*>(search_sha512_kernel), hipFuncCachePreferL1);

  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeLinear;
  resDesc.res.linear.devPtr = dev_countbits;
  resDesc.res.linear.desc.f = hipChannelFormatKindUnsigned;
  resDesc.res.linear.desc.x = 32; // bits per channel
  resDesc.res.linear.sizeInBytes = sizeof(uint32_t)*NUM_COUNTBITS_WORDS;

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.readMode = hipReadModeElementType;
  
  hipCreateTextureObject(&dc_as_tex, &resDesc, &texDesc, NULL);
  /* XXX - WORKING LEFT OFF HERE  */


  return 0;

}

GPUHasher::~GPUHasher() {
  if (dev_hashes != NULL) { hipFree(dev_hashes); }
  if (dev_data != NULL) { hipFree(dev_data); }
}

int GPUHasher::ComputeHashes(uint64_t data[16], uint64_t *hashes) {
  hipError_t error;
  hipStream_t *streamptr = (hipStream_t *)opaqueStream_t;
  uint64_t mydata[16];

  for (int i = 0; i < 16; i++) mydata[i] = data[i];
  for (int i = 1; i < 5; i++) {
    mydata[i] = SWAP64(mydata[i]);
  }

  error = hipMemcpy(dev_data, mydata, sizeof(uint64_t)*16, hipMemcpyHostToDevice);
  if (error != hipSuccess) {
    fprintf(stderr, "Could not memcpy dev_data (%d)\n", error);
    return -1;
  }

  // I want:  64 threads per block
  // 128 blocks per grid entry
  // 1024 grid slots



  dim3 gridsize(2048,32);
  hipMemsetAsync(dev_results, 0, sizeof(uint64_t)*N_RESULTS, *streamptr);
  hipMemsetAsync(dev_countbits, 0, sizeof(uint32_t)*NUM_COUNTBITS_WORDS, *streamptr);
  search_sha512_kernel<<<gridsize, 128, 0, *streamptr>>>(dev_data, dev_hashes, dev_countbits);

  filter_sha512_kernel<<<gridsize, 128, 0, *streamptr>>>(dev_hashes, dev_countbits, dc_as_tex);
  hipMemsetAsync(dev_countbits, 0, sizeof(uint32_t)*NUM_COUNTBITS_WORDS, *streamptr);
  populate_filter_kernel<<<gridsize, 128, 0, *streamptr>>>(dev_hashes, dev_countbits);

  filter_and_rewrite_sha512_kernel<<<gridsize, 128, 0, *streamptr>>>(dev_hashes, dev_countbits, dc_as_tex, dev_results);
  error = hipMemcpyAsync(hashes, dev_results, sizeof(uint64_t)*N_RESULTS, hipMemcpyDeviceToHost, *streamptr);

  error = hipDeviceSynchronize();
  if (error != hipSuccess) {
    fprintf(stderr, "Error in kernel exec (%d)\n", error);
    return -1;
  }

  if (error != hipSuccess) {
    fprintf(stderr, "Could not memcpy dev_hashes out (%d)\n", error);
    return -1;
  }
  return 0;
}

#define SHA512_HASH_WORDS 8 /* 64 bit words */

__constant__ const uint64_t iv512[SHA512_HASH_WORDS] = {
  0x6a09e667f3bcc908LL,
  0xbb67ae8584caa73bLL,
  0x3c6ef372fe94f82bLL,
  0xa54ff53a5f1d36f1LL,
  0x510e527fade682d1LL,
  0x9b05688c2b3e6c1fLL,
  0x1f83d9abfb41bd6bLL,
  0x5be0cd19137e2179LL
};

__device__
void gpu_set_or_double(__restrict__ uint32_t *countbits, uint32_t whichbit) {
  /* Kind of like a saturating add of two bit values.
   * First set is 00 -> 01.  Second set is 01 -> 11
   * Beyond that stays 11
   */
  uint32_t whichword = whichbit/16;
  uint32_t bitpat = 1UL << (2*(whichbit%16));
  uint32_t old = atomicOr(&countbits[whichword], bitpat);
  if (old & bitpat) {
    uint32_t secondbit = (1UL<<((2*(whichbit%16)) +1));
    if (!(old & secondbit)) {
      atomicOr(&countbits[whichword], secondbit);
    }
  }
}

__device__ inline
void gpu_add_to_filter(__restrict__ uint32_t *countbits, const uint64_t hash) {
  uint32_t whichbit = (uint32_t(hash) & ((1UL<<GPU_COUNTBITS_SLOTS_POWER)-1));
  gpu_set_or_double(countbits, whichbit);
}

__device__ inline
bool gpu_is_in_filter_twice(const __restrict__ uint32_t *countbits, const uint64_t hash) {
  uint32_t whichbit = (uint32_t(hash) & ((1UL<<GPU_COUNTBITS_SLOTS_POWER)-1));
  uint32_t cbits = countbits[whichbit/16];
  
  return (cbits & (1UL<<((2*(whichbit%16))+1)));
}

__device__ inline
bool gpu_is_in_filter_twice_tex(hipTextureObject_t countbits, const uint64_t hash) {
  uint32_t whichbit = (uint32_t(hash) & ((1UL<<GPU_COUNTBITS_SLOTS_POWER)-1));
  uint32_t cbits  = tex1Dfetch<unsigned>(countbits, int(whichbit/16));

  return (cbits & (1UL<<((2*(whichbit%16))+1)));
}


__global__
void search_sha512_kernel(const __restrict__ uint64_t *dev_data, __restrict__ uint64_t *dev_hashes, __restrict__ uint32_t *dev_countbits) {
  uint64_t H[8];
  uint64_t D[5];
  uint32_t spot = (((gridDim.x * blockIdx.y) + blockIdx.x)* blockDim.x) + threadIdx.x;
  for (int i = 0; i < 5; i++) {
    D[i] = dev_data[i]; /* constant memory would be better */
  }

  D[0] = (D[0] & 0xffffffff00000000) | (spot*8);

  sha512_block(H, D);

#define POOLSIZE (1<<23)
  for (int i = 0; i < 8; i++) {
    dev_hashes[i*POOLSIZE+spot] = H[i];
  }

  for (int i = 0; i < 8; i++) {
    gpu_add_to_filter(dev_countbits, H[i]);
  }
}

__global__
void filter_sha512_kernel(__restrict__ uint64_t *dev_hashes, const __restrict__ uint32_t *dev_countbits, hipTextureObject_t dc_as_tex) {
  uint32_t spot = (((gridDim.x * blockIdx.y) + blockIdx.x)* blockDim.x) + threadIdx.x;
  for (int i = 0; i < 8; i++) {
    uint64_t myword = dev_hashes[i*POOLSIZE+spot];
    //bool c = gpu_is_in_filter_twice(dev_countbits, myword);
    bool c = gpu_is_in_filter_twice_tex(dc_as_tex, myword);
    if (!c) {
      dev_hashes[i*POOLSIZE+spot] = 0;
    }
  }
}

__global__
void populate_filter_kernel(__restrict__ uint64_t *dev_hashes, __restrict__ uint32_t *dev_countbits) {
  uint32_t spot = (((gridDim.x * blockIdx.y) + blockIdx.x)* blockDim.x) + threadIdx.x;
  for (int i = 0; i < 8; i++) {
    uint64_t myword = dev_hashes[i*POOLSIZE+spot];
    if (myword) {
      gpu_add_to_filter(dev_countbits, (myword>>18));
    }
  }
}

__global__
void filter_and_rewrite_sha512_kernel(__restrict__ uint64_t *dev_hashes, const __restrict__ uint32_t *dev_countbits, hipTextureObject_t dc_as_tex, __restrict__ uint64_t *dev_results) {
  uint32_t spot = (((gridDim.x * blockIdx.y) + blockIdx.x)* blockDim.x) + threadIdx.x;
  for (int i = 0; i < 8; i++) {
    uint64_t myword = dev_hashes[i*POOLSIZE+spot];

    if (myword && gpu_is_in_filter_twice_tex(dc_as_tex, (myword>>18))) {
      uint32_t result_slot = atomicInc((uint32_t *)dev_results, GPUHasher::N_RESULTS);
      dev_results[result_slot*2+1] = (myword);
      dev_results[result_slot*2+2] = (spot*8+i);
    }
  }
}



/***** SHA 512 code is derived from Lukas Odzioba's sha512 crypt implementation within JohnTheRipper.  It has its own copyright */
/*
* This software is Copyright (c) 2011 Lukas Odzioba <lukas dot odzioba at gmail dot com>
* and it is hereby released to the general public under the following terms:
* Redistribution and use in source and binary forms, with or without modification, are permitted.
*/

#define Ch(x,y,z) ((x & y) ^ ( (~x) & z))
#define Maj(x,y,z) ((x & y) ^ (x & z) ^ (y & z))

#define rol(x,n) ((x << n) | (x >> (64-n)))
#define ror(x,n) ((x >> n) | (x << (64-n)))
#define Sigma0(x) ((ror(x,28))  ^ (ror(x,34)) ^ (ror(x,39)))
#define Sigma1(x) ((ror(x,14))  ^ (ror(x,18)) ^ (ror(x,41)))
#define sigma0(x) ((ror(x,1))  ^ (ror(x,8)) ^(x>>7))
#define sigma1(x) ((ror(x,19)) ^ (ror(x,61)) ^(x>>6))



__constant__ uint64_t k[] = {
	0x428a2f98d728ae22LL, 0x7137449123ef65cdLL, 0xb5c0fbcfec4d3b2fLL,
	    0xe9b5dba58189dbbcLL,
	0x3956c25bf348b538LL, 0x59f111f1b605d019LL, 0x923f82a4af194f9bLL,
	    0xab1c5ed5da6d8118LL,
	0xd807aa98a3030242LL, 0x12835b0145706fbeLL, 0x243185be4ee4b28cLL,
	    0x550c7dc3d5ffb4e2LL,
	0x72be5d74f27b896fLL, 0x80deb1fe3b1696b1LL, 0x9bdc06a725c71235LL,
	    0xc19bf174cf692694LL,
	0xe49b69c19ef14ad2LL, 0xefbe4786384f25e3LL, 0x0fc19dc68b8cd5b5LL,
	    0x240ca1cc77ac9c65LL,
	0x2de92c6f592b0275LL, 0x4a7484aa6ea6e483LL, 0x5cb0a9dcbd41fbd4LL,
	    0x76f988da831153b5LL,
	0x983e5152ee66dfabLL, 0xa831c66d2db43210LL, 0xb00327c898fb213fLL,
	    0xbf597fc7beef0ee4LL,
	0xc6e00bf33da88fc2LL, 0xd5a79147930aa725LL, 0x06ca6351e003826fLL,
	    0x142929670a0e6e70LL,
	0x27b70a8546d22ffcLL, 0x2e1b21385c26c926LL, 0x4d2c6dfc5ac42aedLL,
	    0x53380d139d95b3dfLL,
	0x650a73548baf63deLL, 0x766a0abb3c77b2a8LL, 0x81c2c92e47edaee6LL,
	    0x92722c851482353bLL,
	0xa2bfe8a14cf10364LL, 0xa81a664bbc423001LL, 0xc24b8b70d0f89791LL,
	    0xc76c51a30654be30LL,
	0xd192e819d6ef5218LL, 0xd69906245565a910LL, 0xf40e35855771202aLL,
	    0x106aa07032bbd1b8LL,
	0x19a4c116b8d2d0c8LL, 0x1e376c085141ab53LL, 0x2748774cdf8eeb99LL,
	    0x34b0bcb5e19b48a8LL,
	0x391c0cb3c5c95a63LL, 0x4ed8aa4ae3418acbLL, 0x5b9cca4f7763e373LL,
	    0x682e6ff3d6b2b8a3LL,
	0x748f82ee5defb2fcLL, 0x78a5636f43172f60LL, 0x84c87814a1f0ab72LL,
	    0x8cc702081a6439ecLL,
	0x90befffa23631e28LL, 0xa4506cebde82bde9LL, 0xbef9a3f7b2c67915LL,
	    0xc67178f2e372532bLL,
	0xca273eceea26619cLL, 0xd186b8c721c0c207LL, 0xeada7dd6cde0eb1eLL,
	    0xf57d4f7fee6ed178LL,
	0x06f067aa72176fbaLL, 0x0a637dc5a2c898a6LL, 0x113f9804bef90daeLL,
	    0x1b710b35131c471bLL,
	0x28db77f523047d84LL, 0x32caab7b40c72493LL, 0x3c9ebe0a15c9bebcLL,
	    0x431d67c49c100d4cLL,
	0x4cc5d4becb3e42b6LL, 0x597f299cfc657e2aLL, 0x5fcb6fab3ad6faecLL,
	    0x6c44198c4a475817LL,
};

__device__ void sha512_block(uint64_t H[8], const uint64_t data[5])
{

  uint64_t w[16];

  /* If really feel like shaving ops, this could be partially
   * swapped and n swapped in as 32 bits only if desired */
        w[0] = SWAP64(data[0]);

#pragma unroll
	for (int i = 1; i < 5; i++)
		w[i] = data[i];
#pragma unroll
	for (int i = 5; i < 15; i++) {
	  w[i] = 0;
	}
	w[15] = 0x120; /* SWAP64(0x2001000000000000ULL); */

	uint64_t t1, t2;

	/* i = 0 */
	uint64_t g = iv512[5];
        uint64_t e = 0xf7689eb47ab51f91ULL + w[0];
        uint64_t c = iv512[1];
        uint64_t b = iv512[0];
        uint64_t a = 0x954d6b38bcfcddf5ULL + w[0];
        uint64_t f = iv512[4];
	
        /* i=1 */
        t1 = 0x90bb1e3d1f312338ULL + Sigma1(e) + Ch(e, f, g) + w[1];
        t2 = Maj(a, b, c) + Sigma0(a);
        
        g = iv512[4];
        f = e;
        e = iv512[2] + t1;
        uint64_t d = iv512[1];
        c = iv512[0];
        b = a;
        a = t1 + t2;

        /* i=2 */
        t1 = 0x50c6645c178ba74eULL + Sigma1(e) + Ch(e, f, g) + w[2];
        t2 = Maj(a, b, c) + Sigma0(a);
        
        g = f;
        f = e;
        e = iv512[1] + t1;
        d = iv512[0];
        c = b;
        b = a;
        a = t1 + t2;

        /* i=3 */
        t1 = 0x3ac42e252f705e8dULL + w[3] + Sigma1(e) + Ch(e, f, g);
        t2 = Maj(a, b, c) + Sigma0(a);

        uint64_t h = g;
        g = f;
        f = e;
        e = d + t1;
        d = c;
        c = b;
        b = a;
        a = t1 + t2;

        /* i=4 */
        t1 = k[4] + w[4] + h + Sigma1(e) + Ch(e, f, g);
        t2 = Maj(a, b, c) + Sigma0(a);
        
        h = g;
        g = f;
        f = e;
        e = d + t1;
        d = c;
        c = b;
        b = a;
        a = t1 + t2;

	
	/* Unrolled to this point so we can remove w[i] */
#pragma unroll
	for (int i = 5; i < 15; i++) {
		t1 = k[i] + h + Sigma1(e) + Ch(e, f, g);
		t2 = Maj(a, b, c) + Sigma0(a);

		h = g;
		g = f;
		f = e;
		e = d + t1;
		d = c;
		c = b;
		b = a;
		a = t1 + t2;
	}

	t1 = k[15] + w[15] + h + Sigma1(e) + Ch(e, f, g);
	t2 = Maj(a, b, c) + Sigma0(a);
		
	h = g;
	g = f;
	f = e;
	e = d + t1;
	d = c;
	c = b;
	b = a;
	a = t1 + t2;


#pragma unroll
	for (int i = 16; i < 80; i++) {


		w[i & 15] =sigma1(w[(i - 2) & 15]) + sigma0(w[(i - 15) & 15]) + w[(i -16) & 15] + w[(i - 7) & 15];
		t1 = k[i] + w[i & 15] + h + Sigma1(e) + Ch(e, f, g);
		t2 = Maj(a, b, c) + Sigma0(a);

		h = g;
		g = f;
		f = e;
		e = d + t1;
		d = c;
		c = b;
		b = a;
		a = t1 + t2;

	}

        H[0] = iv512[0] + a;
	H[1] = iv512[1] + b;
	H[2] = iv512[2] + c;
	H[3] = iv512[3] + d;
	H[4] = iv512[4] + e;
	H[5] = iv512[5] + f;
	H[6] = iv512[6] + g;
	H[7] = iv512[7] + h;

#if 1
//#pragma unroll
	for (int i = 0; i < 8; i++) {
	  //H[i] = (SWAP64(H[i]));
	  H[i] = (H[i] & 0xc0ffffffffffffULL);
	}
#endif
}
